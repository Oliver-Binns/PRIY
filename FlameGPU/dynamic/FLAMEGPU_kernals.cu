#include "hip/hip_runtime.h"


/*
* FLAME GPU v 1.4.0 for CUDA 6
* Copyright 2015 University of Sheffield.
* Author: Dr Paul Richmond
* Contact: p.richmond@sheffield.ac.uk (http://www.paulrichmond.staff.shef.ac.uk)
*
* University of Sheffield retain all intellectual property and
* proprietary rights in and to this software and related documentation.
* Any use, reproduction, disclosure, or distribution of this software
* and related documentation without an express license agreement from
* University of Sheffield is strictly prohibited.
*
* For terms of licence agreement please attached licence or view licence
* on www.flamegpu.com website.
*
*/

#ifndef _FLAMEGPU_KERNELS_H_
#define _FLAMEGPU_KERNELS_H_

#include "header.h"


/* Agent count constants */

__constant__ int d_xmachine_memory_LTin_count;

__constant__ int d_xmachine_memory_LTi_count;

__constant__ int d_xmachine_memory_LTo_count;

/* Agent state count constants */

__constant__ int d_xmachine_memory_LTin_ltin_random_movement_count;

__constant__ int d_xmachine_memory_LTin_stable_contact_count;

__constant__ int d_xmachine_memory_LTin_localised_movement_count;

__constant__ int d_xmachine_memory_LTi_lti_random_movement_count;

__constant__ int d_xmachine_memory_LTi_responding_count;

__constant__ int d_xmachine_memory_LTi_contact_count;

__constant__ int d_xmachine_memory_LTi_adhesion_count;

__constant__ int d_xmachine_memory_LTo_no_expression_count;

__constant__ int d_xmachine_memory_LTo_expression_count;

__constant__ int d_xmachine_memory_LTo_adhesion_upregulation_count;

__constant__ int d_xmachine_memory_LTo_chemokine_upregulation_count;

__constant__ int d_xmachine_memory_LTo_mature_count;

__constant__ int d_xmachine_memory_LTo_downregulated_count;


/* Message constants */

/* location Message variables */
/* Non partitioned and spatial partitioned message variables  */
__constant__ int d_message_location_count;         /**< message list counter*/
__constant__ int d_message_location_output_type;   /**< message output type (single or optional)*/

	
    
//include each function file

#include "functions.c"
    
/* Texture bindings */

    
#define WRAP(x,m) (((x)<m)?(x):(x%m)) /**< Simple wrap */
#define sWRAP(x,m) (((x)<m)?(((x)<0)?(m+(x)):(x)):(m-(x))) /**<signed integer wrap (no modulus) for negatives where 2m > |x| > m */

//PADDING WILL ONLY AVOID SM CONFLICTS FOR 32BIT
//SM_OFFSET REQUIRED AS FERMI STARTS INDEXING MEMORY FROM LOCATION 0 (i.e. NULL)??
__constant__ int d_SM_START;
__constant__ int d_PADDING;

//SM addressing macro to avoid conflicts (32 bit only)
#define SHARE_INDEX(i, s) ((((s) + d_PADDING)* (i))+d_SM_START) /**<offset struct size by padding to avoid bank conflicts */

//if doubel support is needed then define the following function which requires sm_13 or later
#ifdef _DOUBLE_SUPPORT_REQUIRED_
__inline__ __device__ double tex1DfetchDouble(texture<int2, 1, hipReadModeElementType> tex, int i)
{
	int2 v = tex1Dfetch(tex, i);
  //IF YOU HAVE AN ERROR HERE THEN YOU ARE USING DOUBLE VALUES IN AGENT MEMORY AND NOT COMPILING FOR DOUBLE SUPPORTED HARDWARE
  //To compile for double supported hardware change the CUDA Build rule property "Use sm_13 Architecture (double support)" on the CUDA-Specific Propert Page of the CUDA Build Rule for simulation.cu
	return __hiloint2double(v.y, v.x);
}
#endif

/* Helper functions */
/** next_cell
 * Function used for finding the next cell when using spatial partitioning
 * Upddates the relative cell variable which can have value of -1, 0 or +1
 * @param relative_cell pointer to the relative cell position
 * @return boolean if there is a next cell. True unless relative_Cell value was 1,1,1
 */
__device__ int next_cell3D(glm::ivec3* relative_cell)
{
	if (relative_cell->x < 1)
	{
		relative_cell->x++;
		return true;
	}
	relative_cell->x = -1;

	if (relative_cell->y < 1)
	{
		relative_cell->y++;
		return true;
	}
	relative_cell->y = -1;
	
	if (relative_cell->z < 1)
	{
		relative_cell->z++;
		return true;
	}
	relative_cell->z = -1;
	
	return false;
}

/** next_cell2D
 * Function used for finding the next cell when using spatial partitioning. Z component is ignored
 * Upddates the relative cell variable which can have value of -1, 0 or +1
 * @param relative_cell pointer to the relative cell position
 * @return boolean if there is a next cell. True unless relative_Cell value was 1,1
 */
__device__ int next_cell2D(glm::ivec3* relative_cell)
{
	if (relative_cell->x < 1)
	{
		relative_cell->x++;
		return true;
	}
	relative_cell->x = -1;

	if (relative_cell->y < 1)
	{
		relative_cell->y++;
		return true;
	}
	relative_cell->y = -1;
	
	return false;
}


////////////////////////////////////////////////////////////////////////////////////////////////////////
/* Dyanamically created LTin agent functions */

/** reset_LTin_scan_input
 * LTin agent reset scan input function
 * @param agents The xmachine_memory_LTin_list agent list
 */
__global__ void reset_LTin_scan_input(xmachine_memory_LTin_list* agents){

	//global thread index
	int index = (blockIdx.x*blockDim.x) + threadIdx.x;

	agents->_position[index] = 0;
	agents->_scan_input[index] = 0;
}



/** scatter_LTin_Agents
 * LTin scatter agents function (used after agent birth/death)
 * @param agents_dst xmachine_memory_LTin_list agent list destination
 * @param agents_src xmachine_memory_LTin_list agent list source
 * @param dst_agent_count index to start scattering agents from
 */
__global__ void scatter_LTin_Agents(xmachine_memory_LTin_list* agents_dst, xmachine_memory_LTin_list* agents_src, int dst_agent_count, int number_to_scatter){
	//global thread index
	int index = (blockIdx.x*blockDim.x) + threadIdx.x;

	int _scan_input = agents_src->_scan_input[index];

	//if optional message is to be written. 
	//must check agent is within number to scatter as unused threads may have scan input = 1
	if ((_scan_input == 1)&&(index < number_to_scatter)){
		int output_index = agents_src->_position[index] + dst_agent_count;

		//AoS - xmachine_message_location Un-Coalesced scattered memory write     
        agents_dst->_position[output_index] = output_index;        
		agents_dst->x[output_index] = agents_src->x[index];        
		agents_dst->y[output_index] = agents_src->y[index];
	}
}

/** append_LTin_Agents
 * LTin scatter agents function (used after agent birth/death)
 * @param agents_dst xmachine_memory_LTin_list agent list destination
 * @param agents_src xmachine_memory_LTin_list agent list source
 * @param dst_agent_count index to start scattering agents from
 */
__global__ void append_LTin_Agents(xmachine_memory_LTin_list* agents_dst, xmachine_memory_LTin_list* agents_src, int dst_agent_count, int number_to_append){
	//global thread index
	int index = (blockIdx.x*blockDim.x) + threadIdx.x;

	//must check agent is within number to append as unused threads may have scan input = 1
    if (index < number_to_append){
	    int output_index = index + dst_agent_count;

	    //AoS - xmachine_message_location Un-Coalesced scattered memory write
	    agents_dst->_position[output_index] = output_index;
	    agents_dst->x[output_index] = agents_src->x[index];
	    agents_dst->y[output_index] = agents_src->y[index];
    }
}

/** add_LTin_agent
 * Continuous LTin agent add agent function writes agent data to agent swap
 * @param agents xmachine_memory_LTin_list to add agents to 
 * @param x agent variable of type float
 * @param y agent variable of type float
 */
template <int AGENT_TYPE>
__device__ void add_LTin_agent(xmachine_memory_LTin_list* agents, float x, float y){
	
	int index;
    
    //calculate the agents index in global agent list (depends on agent type)
	if (AGENT_TYPE == DISCRETE_2D){
		int width = (blockDim.x* gridDim.x);
		glm::ivec2 global_position;
		global_position.x = (blockIdx.x*blockDim.x) + threadIdx.x;
		global_position.y = (blockIdx.y*blockDim.y) + threadIdx.y;
		index = global_position.x + (global_position.y* width);
	}else//AGENT_TYPE == CONTINOUS
		index = threadIdx.x + blockIdx.x*blockDim.x;

	//for prefix sum
	agents->_position[index] = 0;
	agents->_scan_input[index] = 1;

	//write data to new buffer
	agents->x[index] = x;
	agents->y[index] = y;

}

//non templated version assumes DISCRETE_2D but works also for CONTINUOUS
__device__ void add_LTin_agent(xmachine_memory_LTin_list* agents, float x, float y){
    add_LTin_agent<DISCRETE_2D>(agents, x, y);
}

/** reorder_LTin_agents
 * Continuous LTin agent areorder function used after key value pairs have been sorted
 * @param values sorted index values
 * @param unordered_agents list of unordered agents
 * @ param ordered_agents list used to output ordered agents
 */
__global__ void reorder_LTin_agents(unsigned int* values, xmachine_memory_LTin_list* unordered_agents, xmachine_memory_LTin_list* ordered_agents)
{
	int index = (blockIdx.x*blockDim.x) + threadIdx.x;

	uint old_pos = values[index];

	//reorder agent data
	ordered_agents->x[index] = unordered_agents->x[old_pos];
	ordered_agents->y[index] = unordered_agents->y[old_pos];
}

////////////////////////////////////////////////////////////////////////////////////////////////////////
/* Dyanamically created LTi agent functions */

/** reset_LTi_scan_input
 * LTi agent reset scan input function
 * @param agents The xmachine_memory_LTi_list agent list
 */
__global__ void reset_LTi_scan_input(xmachine_memory_LTi_list* agents){

	//global thread index
	int index = (blockIdx.x*blockDim.x) + threadIdx.x;

	agents->_position[index] = 0;
	agents->_scan_input[index] = 0;
}



/** scatter_LTi_Agents
 * LTi scatter agents function (used after agent birth/death)
 * @param agents_dst xmachine_memory_LTi_list agent list destination
 * @param agents_src xmachine_memory_LTi_list agent list source
 * @param dst_agent_count index to start scattering agents from
 */
__global__ void scatter_LTi_Agents(xmachine_memory_LTi_list* agents_dst, xmachine_memory_LTi_list* agents_src, int dst_agent_count, int number_to_scatter){
	//global thread index
	int index = (blockIdx.x*blockDim.x) + threadIdx.x;

	int _scan_input = agents_src->_scan_input[index];

	//if optional message is to be written. 
	//must check agent is within number to scatter as unused threads may have scan input = 1
	if ((_scan_input == 1)&&(index < number_to_scatter)){
		int output_index = agents_src->_position[index] + dst_agent_count;

		//AoS - xmachine_message_location Un-Coalesced scattered memory write     
        agents_dst->_position[output_index] = output_index;        
		agents_dst->x[output_index] = agents_src->x[index];        
		agents_dst->y[output_index] = agents_src->y[index];
	}
}

/** append_LTi_Agents
 * LTi scatter agents function (used after agent birth/death)
 * @param agents_dst xmachine_memory_LTi_list agent list destination
 * @param agents_src xmachine_memory_LTi_list agent list source
 * @param dst_agent_count index to start scattering agents from
 */
__global__ void append_LTi_Agents(xmachine_memory_LTi_list* agents_dst, xmachine_memory_LTi_list* agents_src, int dst_agent_count, int number_to_append){
	//global thread index
	int index = (blockIdx.x*blockDim.x) + threadIdx.x;

	//must check agent is within number to append as unused threads may have scan input = 1
    if (index < number_to_append){
	    int output_index = index + dst_agent_count;

	    //AoS - xmachine_message_location Un-Coalesced scattered memory write
	    agents_dst->_position[output_index] = output_index;
	    agents_dst->x[output_index] = agents_src->x[index];
	    agents_dst->y[output_index] = agents_src->y[index];
    }
}

/** add_LTi_agent
 * Continuous LTi agent add agent function writes agent data to agent swap
 * @param agents xmachine_memory_LTi_list to add agents to 
 * @param x agent variable of type float
 * @param y agent variable of type float
 */
template <int AGENT_TYPE>
__device__ void add_LTi_agent(xmachine_memory_LTi_list* agents, float x, float y){
	
	int index;
    
    //calculate the agents index in global agent list (depends on agent type)
	if (AGENT_TYPE == DISCRETE_2D){
		int width = (blockDim.x* gridDim.x);
		glm::ivec2 global_position;
		global_position.x = (blockIdx.x*blockDim.x) + threadIdx.x;
		global_position.y = (blockIdx.y*blockDim.y) + threadIdx.y;
		index = global_position.x + (global_position.y* width);
	}else//AGENT_TYPE == CONTINOUS
		index = threadIdx.x + blockIdx.x*blockDim.x;

	//for prefix sum
	agents->_position[index] = 0;
	agents->_scan_input[index] = 1;

	//write data to new buffer
	agents->x[index] = x;
	agents->y[index] = y;

}

//non templated version assumes DISCRETE_2D but works also for CONTINUOUS
__device__ void add_LTi_agent(xmachine_memory_LTi_list* agents, float x, float y){
    add_LTi_agent<DISCRETE_2D>(agents, x, y);
}

/** reorder_LTi_agents
 * Continuous LTi agent areorder function used after key value pairs have been sorted
 * @param values sorted index values
 * @param unordered_agents list of unordered agents
 * @ param ordered_agents list used to output ordered agents
 */
__global__ void reorder_LTi_agents(unsigned int* values, xmachine_memory_LTi_list* unordered_agents, xmachine_memory_LTi_list* ordered_agents)
{
	int index = (blockIdx.x*blockDim.x) + threadIdx.x;

	uint old_pos = values[index];

	//reorder agent data
	ordered_agents->x[index] = unordered_agents->x[old_pos];
	ordered_agents->y[index] = unordered_agents->y[old_pos];
}

////////////////////////////////////////////////////////////////////////////////////////////////////////
/* Dyanamically created LTo agent functions */

/** reset_LTo_scan_input
 * LTo agent reset scan input function
 * @param agents The xmachine_memory_LTo_list agent list
 */
__global__ void reset_LTo_scan_input(xmachine_memory_LTo_list* agents){

	//global thread index
	int index = (blockIdx.x*blockDim.x) + threadIdx.x;

	agents->_position[index] = 0;
	agents->_scan_input[index] = 0;
}



/** scatter_LTo_Agents
 * LTo scatter agents function (used after agent birth/death)
 * @param agents_dst xmachine_memory_LTo_list agent list destination
 * @param agents_src xmachine_memory_LTo_list agent list source
 * @param dst_agent_count index to start scattering agents from
 */
__global__ void scatter_LTo_Agents(xmachine_memory_LTo_list* agents_dst, xmachine_memory_LTo_list* agents_src, int dst_agent_count, int number_to_scatter){
	//global thread index
	int index = (blockIdx.x*blockDim.x) + threadIdx.x;

	int _scan_input = agents_src->_scan_input[index];

	//if optional message is to be written. 
	//must check agent is within number to scatter as unused threads may have scan input = 1
	if ((_scan_input == 1)&&(index < number_to_scatter)){
		int output_index = agents_src->_position[index] + dst_agent_count;

		//AoS - xmachine_message_location Un-Coalesced scattered memory write     
        agents_dst->_position[output_index] = output_index;        
		agents_dst->x[output_index] = agents_src->x[index];        
		agents_dst->y[output_index] = agents_src->y[index];
	}
}

/** append_LTo_Agents
 * LTo scatter agents function (used after agent birth/death)
 * @param agents_dst xmachine_memory_LTo_list agent list destination
 * @param agents_src xmachine_memory_LTo_list agent list source
 * @param dst_agent_count index to start scattering agents from
 */
__global__ void append_LTo_Agents(xmachine_memory_LTo_list* agents_dst, xmachine_memory_LTo_list* agents_src, int dst_agent_count, int number_to_append){
	//global thread index
	int index = (blockIdx.x*blockDim.x) + threadIdx.x;

	//must check agent is within number to append as unused threads may have scan input = 1
    if (index < number_to_append){
	    int output_index = index + dst_agent_count;

	    //AoS - xmachine_message_location Un-Coalesced scattered memory write
	    agents_dst->_position[output_index] = output_index;
	    agents_dst->x[output_index] = agents_src->x[index];
	    agents_dst->y[output_index] = agents_src->y[index];
    }
}

/** add_LTo_agent
 * Continuous LTo agent add agent function writes agent data to agent swap
 * @param agents xmachine_memory_LTo_list to add agents to 
 * @param x agent variable of type float
 * @param y agent variable of type float
 */
template <int AGENT_TYPE>
__device__ void add_LTo_agent(xmachine_memory_LTo_list* agents, float x, float y){
	
	int index;
    
    //calculate the agents index in global agent list (depends on agent type)
	if (AGENT_TYPE == DISCRETE_2D){
		int width = (blockDim.x* gridDim.x);
		glm::ivec2 global_position;
		global_position.x = (blockIdx.x*blockDim.x) + threadIdx.x;
		global_position.y = (blockIdx.y*blockDim.y) + threadIdx.y;
		index = global_position.x + (global_position.y* width);
	}else//AGENT_TYPE == CONTINOUS
		index = threadIdx.x + blockIdx.x*blockDim.x;

	//for prefix sum
	agents->_position[index] = 0;
	agents->_scan_input[index] = 1;

	//write data to new buffer
	agents->x[index] = x;
	agents->y[index] = y;

}

//non templated version assumes DISCRETE_2D but works also for CONTINUOUS
__device__ void add_LTo_agent(xmachine_memory_LTo_list* agents, float x, float y){
    add_LTo_agent<DISCRETE_2D>(agents, x, y);
}

/** reorder_LTo_agents
 * Continuous LTo agent areorder function used after key value pairs have been sorted
 * @param values sorted index values
 * @param unordered_agents list of unordered agents
 * @ param ordered_agents list used to output ordered agents
 */
__global__ void reorder_LTo_agents(unsigned int* values, xmachine_memory_LTo_list* unordered_agents, xmachine_memory_LTo_list* ordered_agents)
{
	int index = (blockIdx.x*blockDim.x) + threadIdx.x;

	uint old_pos = values[index];

	//reorder agent data
	ordered_agents->x[index] = unordered_agents->x[old_pos];
	ordered_agents->y[index] = unordered_agents->y[old_pos];
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
/* Dyanamically created location message functions */


/** add_location_message
 * Add non partitioned or spatially partitioned location message
 * @param messages xmachine_message_location_list message list to add too
 * @param x agent variable of type float
 * @param y agent variable of type float
 */
__device__ void add_location_message(xmachine_message_location_list* messages, float x, float y){

	//global thread index
	int index = (blockIdx.x*blockDim.x) + threadIdx.x + d_message_location_count;

	int _position;
	int _scan_input;

	//decide output position
	if(d_message_location_output_type == single_message){
		_position = index; //same as agent position
		_scan_input = 0;
	}else if (d_message_location_output_type == optional_message){
		_position = 0;	   //to be calculated using Prefix sum
		_scan_input = 1;
	}

	//AoS - xmachine_message_location Coalesced memory write
	messages->_scan_input[index] = _scan_input;	
	messages->_position[index] = _position;
	messages->x[index] = x;
	messages->y[index] = y;

}

/**
 * Scatter non partitioned or spatially partitioned location message (for optional messages)
 * @param messages scatter_optional_location_messages Sparse xmachine_message_location_list message list
 * @param message_swap temp xmachine_message_location_list message list to scatter sparse messages to
 */
__global__ void scatter_optional_location_messages(xmachine_message_location_list* messages, xmachine_message_location_list* messages_swap){
	//global thread index
	int index = (blockIdx.x*blockDim.x) + threadIdx.x;

	int _scan_input = messages_swap->_scan_input[index];

	//if optional message is to be written
	if (_scan_input == 1){
		int output_index = messages_swap->_position[index] + d_message_location_count;

		//AoS - xmachine_message_location Un-Coalesced scattered memory write
		messages->_position[output_index] = output_index;
		messages->x[output_index] = messages_swap->x[index];
		messages->y[output_index] = messages_swap->y[index];				
	}
}

/** reset_location_swaps
 * Reset non partitioned or spatially partitioned location message swaps (for scattering optional messages)
 * @param message_swap message list to reset _position and _scan_input values back to 0
 */
__global__ void reset_location_swaps(xmachine_message_location_list* messages_swap){

	//global thread index
	int index = (blockIdx.x*blockDim.x) + threadIdx.x;

	messages_swap->_position[index] = 0;
	messages_swap->_scan_input[index] = 0;
}

/* Message functions */

__device__ xmachine_message_location* get_first_location_message(xmachine_message_location_list* messages){

	extern __shared__ int sm_data [];
	char* message_share = (char*)&sm_data[0];
	
	//wrap size is the number of tiles required to load all messages
	int wrap_size = (ceil((float)d_message_location_count/ blockDim.x)* blockDim.x);

	//if no messages then return false
	if (wrap_size == 0)
		return false;

	//global thread index
	int global_index = (blockIdx.x*blockDim.x) + threadIdx.x;

	//global thread index
	int index = WRAP(global_index, wrap_size);

	//SoA to AoS - xmachine_message_location Coalesced memory read
	xmachine_message_location temp_message;
	temp_message._position = messages->_position[index];
	temp_message.x = messages->x[index];
	temp_message.y = messages->y[index];

	//AoS to shared memory
	int message_index = SHARE_INDEX(threadIdx.y*blockDim.x+threadIdx.x, sizeof(xmachine_message_location));
	xmachine_message_location* sm_message = ((xmachine_message_location*)&message_share[message_index]);
	sm_message[0] = temp_message;

	__syncthreads();

  //HACK FOR 64 bit addressing issue in sm
	return ((xmachine_message_location*)&message_share[d_SM_START]);
}

__device__ xmachine_message_location* get_next_location_message(xmachine_message_location* message, xmachine_message_location_list* messages){

	extern __shared__ int sm_data [];
	char* message_share = (char*)&sm_data[0];
	
	//wrap size is the number of tiles required to load all messages
	int wrap_size = ceil((float)d_message_location_count/ blockDim.x)*blockDim.x;

	int i = WRAP((message->_position + 1),wrap_size);

	//If end of messages (last message not multiple of gridsize) go to 0 index
	if (i >= d_message_location_count)
		i = 0;

	//Check if back to start position of first message
	if (i == WRAP((blockDim.x* blockIdx.x), wrap_size))
		return false;

	int tile = floor((float)i/(blockDim.x)); //tile is round down position over blockDim
	i = i % blockDim.x;						 //mod i for shared memory index

	//if count == Block Size load next tile int shared memory values
	if (i == 0){
		__syncthreads();					//make sure we don't change shared memory until all threads are here (important for emu-debug mode)
		
		//SoA to AoS - xmachine_message_location Coalesced memory read
		int index = (tile* blockDim.x) + threadIdx.x;
		xmachine_message_location temp_message;
		temp_message._position = messages->_position[index];
		temp_message.x = messages->x[index];
		temp_message.y = messages->y[index];

		//AoS to shared memory
		int message_index = SHARE_INDEX(threadIdx.y*blockDim.x+threadIdx.x, sizeof(xmachine_message_location));
		xmachine_message_location* sm_message = ((xmachine_message_location*)&message_share[message_index]);
		sm_message[0] = temp_message;

		__syncthreads();					//make sure we don't start returning messages until all threads have updated shared memory
	}

	int message_index = SHARE_INDEX(i, sizeof(xmachine_message_location));
	return ((xmachine_message_location*)&message_share[message_index]);
}


	
/////////////////////////////////////////////////////////////////////////////////////////////////////////
/* Dynamically created GPU kernels  */



/**
 *
 */
__global__ void GPUFLAME_ltin_random_move(xmachine_memory_LTin_list* agents, xmachine_message_location_list* location_messages){
	
	//continuous agent: index is agent position in 1D agent list
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  
    //For agents not using non partitioned message input check the agent bounds
    if (index >= d_xmachine_memory_LTin_count)
        return;
    

	//SoA to AoS - xmachine_memory_ltin_random_move Coalesced memory read (arrays point to first item for agent index)
	xmachine_memory_LTin agent;
	agent.x = agents->x[index];
	agent.y = agents->y[index];

	//FLAME function call
	int dead = !ltin_random_move(&agent, location_messages	);
	
	//continuous agent: set reallocation flag
	agents->_scan_input[index]  = dead; 

	//AoS to SoA - xmachine_memory_ltin_random_move Coalesced memory write (ignore arrays)
	agents->x[index] = agent.x;
	agents->y[index] = agent.y;
}

/**
 *
 */
__global__ void GPUFLAME_lti_random_move(xmachine_memory_LTi_list* agents, xmachine_message_location_list* location_messages){
	
	//continuous agent: index is agent position in 1D agent list
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  
    //For agents not using non partitioned message input check the agent bounds
    if (index >= d_xmachine_memory_LTi_count)
        return;
    

	//SoA to AoS - xmachine_memory_lti_random_move Coalesced memory read (arrays point to first item for agent index)
	xmachine_memory_LTi agent;
	agent.x = agents->x[index];
	agent.y = agents->y[index];

	//FLAME function call
	int dead = !lti_random_move(&agent, location_messages	);
	
	//continuous agent: set reallocation flag
	agents->_scan_input[index]  = dead; 

	//AoS to SoA - xmachine_memory_lti_random_move Coalesced memory write (ignore arrays)
	agents->x[index] = agent.x;
	agents->y[index] = agent.y;
}

	
	
/////////////////////////////////////////////////////////////////////////////////////////////////////////////////
/* Rand48 functions */

__device__ static glm::uvec2 RNG_rand48_iterate_single(glm::uvec2 Xn, glm::uvec2 A, glm::uvec2 C)
{
	unsigned int R0, R1;

	// low 24-bit multiplication
	const unsigned int lo00 = __umul24(Xn.x, A.x);
	const unsigned int hi00 = __umulhi(Xn.x, A.x);

	// 24bit distribution of 32bit multiplication results
	R0 = (lo00 & 0xFFFFFF);
	R1 = (lo00 >> 24) | (hi00 << 8);

	R0 += C.x; R1 += C.y;

	// transfer overflows
	R1 += (R0 >> 24);
	R0 &= 0xFFFFFF;

	// cross-terms, low/hi 24-bit multiplication
	R1 += __umul24(Xn.y, A.x);
	R1 += __umul24(Xn.x, A.y);

	R1 &= 0xFFFFFF;

	return glm::uvec2(R0, R1);
}

//Templated function
template <int AGENT_TYPE>
__device__ float rnd(RNG_rand48* rand48){

	int index;
	
	//calculate the agents index in global agent list
	if (AGENT_TYPE == DISCRETE_2D){
		int width = (blockDim.x * gridDim.x);
		glm::ivec2 global_position;
		global_position.x = (blockIdx.x * blockDim.x) + threadIdx.x;
		global_position.y = (blockIdx.y * blockDim.y) + threadIdx.y;
		index = global_position.x + (global_position.y * width);
	}else//AGENT_TYPE == CONTINOUS
		index = threadIdx.x + blockIdx.x*blockDim.x;

	glm::uvec2 state = rand48->seeds[index];
	glm::uvec2 A = rand48->A;
	glm::uvec2 C = rand48->C;

	int rand = ( state.x >> 17 ) | ( state.y << 7);

	// this actually iterates the RNG
	state = RNG_rand48_iterate_single(state, A, C);

	rand48->seeds[index] = state;

	return (float)rand/2147483647;
}

__device__ float rnd(RNG_rand48* rand48){
	return rnd<DISCRETE_2D>(rand48);
}

#endif //_FLAMEGPU_KERNELS_H_
