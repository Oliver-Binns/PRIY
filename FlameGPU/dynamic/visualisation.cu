#include "hip/hip_runtime.h"

/*
* FLAME GPU v 1.4.0 for CUDA 6
* Copyright 2015 University of Sheffield.
* Author: Dr Paul Richmond
* Contact: p.richmond@sheffield.ac.uk (http://www.paulrichmond.staff.shef.ac.uk)
*
* University of Sheffield retain all intellectual property and
* proprietary rights in and to this software and related documentation.
* Any use, reproduction, disclosure, or distribution of this software
* and related documentation without an express license agreement from
* University of Sheffield is strictly prohibited.
*
* For terms of licence agreement please attached licence or view licence
* on www.flamegpu.com website.
*
*/

// includes, project
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <cmath>

#include <GL/glew.h>
#include <GL/glut.h>
#include <cuda_gl_interop.h>
	    
#include "header.h"
#include "visualisation.h"

// bo variables
GLuint sphereVerts;
GLuint sphereNormals;

//Simulation output buffers/textures

GLuint LTin_ltin_random_movement_tbo;
GLuint LTin_ltin_random_movement_displacementTex;

GLuint LTin_stable_contact_tbo;
GLuint LTin_stable_contact_displacementTex;

GLuint LTin_localised_movement_tbo;
GLuint LTin_localised_movement_displacementTex;

GLuint LTi_lti_random_movement_tbo;
GLuint LTi_lti_random_movement_displacementTex;

GLuint LTi_responding_tbo;
GLuint LTi_responding_displacementTex;

GLuint LTi_contact_tbo;
GLuint LTi_contact_displacementTex;

GLuint LTi_adhesion_tbo;
GLuint LTi_adhesion_displacementTex;

GLuint LTo_no_expression_tbo;
GLuint LTo_no_expression_displacementTex;

GLuint LTo_expression_tbo;
GLuint LTo_expression_displacementTex;

GLuint LTo_adhesion_upregulation_tbo;
GLuint LTo_adhesion_upregulation_displacementTex;

GLuint LTo_chemokine_upregulation_tbo;
GLuint LTo_chemokine_upregulation_displacementTex;

GLuint LTo_mature_tbo;
GLuint LTo_mature_displacementTex;

GLuint LTo_downregulated_tbo;
GLuint LTo_downregulated_displacementTex;


// mouse controls
int mouse_old_x, mouse_old_y;
int mouse_buttons = 0;
float rotate_x = 0.0, rotate_y = 0.0;
float translate_z = -VIEW_DISTANCE;

// vertex Shader
GLuint vertexShader;
GLuint fragmentShader;
GLuint shaderProgram;
GLuint vs_displacementMap;
GLuint vs_mapIndex;



//timer
hipEvent_t start, stop;
const int display_rate = 50;
int frame_count;
float frame_time = 0.0;

#ifdef SIMULATION_DELAY
//delay
int delay_count = 0;
#endif

// prototypes
int initGL();
void initShader();
void createVBO( GLuint* vbo, GLuint size);
void deleteVBO( GLuint* vbo);
void createTBO( GLuint* tbo, GLuint* tex, GLuint size);
void deleteTBO( GLuint* tbo);
void setVertexBufferData();
void display();
void keyboard( unsigned char key, int x, int y);
void mouse(int button, int state, int x, int y);
void motion(int x, int y);
void runCuda();
void checkGLError();

/* Error check function for safe CUDA API calling */
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

/* Error check function for post CUDA Kernel calling */
#define gpuErrchkLaunch() { gpuLaunchAssert(__FILE__, __LINE__); }
inline void gpuLaunchAssert(const char *file, int line, bool abort=true)
{
	gpuAssert( hipPeekAtLastError(), file, line );
#ifdef _DEBUG
	gpuAssert( hipDeviceSynchronize(), file, line );
#endif
   
}

const char vertexShaderSource[] = 
{  
	"#extension GL_EXT_gpu_shader4 : enable										\n"
	"uniform samplerBuffer displacementMap;										\n"
	"attribute in float mapIndex;												\n"
	"varying vec3 normal, lightDir;												\n"
	"varying vec4 colour;														\n"
    "void main()																\n"
    "{																			\n"
	"	vec4 position = gl_Vertex;											    \n"
	"	vec4 lookup = texelFetchBuffer(displacementMap, (int)mapIndex);		    \n"
    "	if (lookup.w > 6.5)	                								\n"
	"		colour = vec4(1.0, 1.0, 1.0, 0.0);								    \n"
    "	else if (lookup.w > 5.5)	                								\n"
	"		colour = vec4(1.0, 0.0, 1.0, 0.0);								    \n"
	"	else if (lookup.w > 4.5)	                								\n"
	"		colour = vec4(0.0, 1.0, 1.0, 0.0);								    \n"
    "	else if (lookup.w > 3.5)	                								\n"
	"		colour = vec4(1.0, 1.0, 0.0, 0.0);								    \n"
	"	else if (lookup.w > 2.5)	                								\n"
	"		colour = vec4(0.0, 0.0, 1.0, 0.0);								    \n"
	"	else if (lookup.w > 1.5)	                								\n"
	"		colour = vec4(0.0, 1.0, 0.0, 0.0);								    \n"
    "	else if (lookup.w > 0.5)	                								\n"
	"		colour = vec4(1.0, 0.0, 0.0, 0.0);								    \n"
    "	else                      	                								\n"
	"		colour = vec4(0.0, 0.0, 0.0, 0.0);								    \n"
	"																    		\n"
	"	lookup.w = 1.0;												    		\n"
	"	position += lookup;											    		\n"
	"   gl_Position = gl_ModelViewProjectionMatrix * position;		    		\n"
	"																			\n"
	"	vec3 mvVertex = vec3(gl_ModelViewMatrix * position);			    	\n"
	"	lightDir = vec3(gl_LightSource[0].position.xyz - mvVertex);				\n"
	"	normal = gl_NormalMatrix * gl_Normal;									\n"
    "}																			\n"
};

const char fragmentShaderSource[] = 
{  
	"varying vec3 normal, lightDir;												\n"
	"varying vec4 colour;														\n"
	"void main (void)															\n"
	"{																			\n"
	"	// Defining The Material Colors											\n"
	"	vec4 AmbientColor = vec4(0.25, 0.0, 0.0, 1.0);					\n"
	"	vec4 DiffuseColor = colour;					                	\n"
	"																			\n"
	"	// Scaling The Input Vector To Length 1									\n"
	"	vec3 n_normal = normalize(normal);							        	\n"
	"	vec3 n_lightDir = normalize(lightDir);	                                \n"
	"																			\n"
	"	// Calculating The Diffuse Term And Clamping It To [0;1]				\n"
	"	float DiffuseTerm = clamp(dot(n_normal, n_lightDir), 0.0, 1.0);\n"
	"																			\n"
	"	// Calculating The Final Color											\n"
	"	gl_FragColor = AmbientColor + DiffuseColor * DiffuseTerm;				\n"
	"																			\n"
	"}																			\n"
};

//GPU Kernels

__global__ void output_LTin_agent_to_VBO(xmachine_memory_LTin_list* agents, glm::vec4* vbo, glm::vec3 centralise){

	//global thread index
	int index = __mul24(blockIdx.x,blockDim.x) + threadIdx.x;

	vbo[index].x = 0.0;
	vbo[index].y = 0.0;
	vbo[index].z = 0.0;
	
	vbo[index].x = agents->x[index] - centralise.x;
	vbo[index].y = agents->y[index] - centralise.y;
	vbo[index].z = 0.0;
	vbo[index].w = 1.0;
}

__global__ void output_LTi_agent_to_VBO(xmachine_memory_LTi_list* agents, glm::vec4* vbo, glm::vec3 centralise){

	//global thread index
	int index = __mul24(blockIdx.x,blockDim.x) + threadIdx.x;

	vbo[index].x = 0.0;
	vbo[index].y = 0.0;
	vbo[index].z = 0.0;
	
	vbo[index].x = agents->x[index] - centralise.x;
	vbo[index].y = agents->y[index] - centralise.y;
	vbo[index].z = 0.0;
	vbo[index].w = 1.0;
}

__global__ void output_LTo_agent_to_VBO(xmachine_memory_LTo_list* agents, glm::vec4* vbo, glm::vec3 centralise){

	//global thread index
	int index = __mul24(blockIdx.x,blockDim.x) + threadIdx.x;

	vbo[index].x = 0.0;
	vbo[index].y = 0.0;
	vbo[index].z = 0.0;
	
	vbo[index].x = agents->x[index] - centralise.x;
	vbo[index].y = agents->y[index] - centralise.y;
	vbo[index].z = 0.0;
	vbo[index].w = 1.0;
}


void initVisualisation()
{
	//set the CUDA GL device: Will cause an error without this since CUDA 3.0
	cudaGLSetGLDevice(0);

	// Create GL context
	int   argc   = 1;
	char *argv[] = {"GLUT application", NULL};
	glutInit( &argc, argv);
	glutInitDisplayMode( GLUT_RGBA | GLUT_DOUBLE);
	glutInitWindowSize( WINDOW_WIDTH, WINDOW_HEIGHT);
	glutCreateWindow( "FLAME GPU Visualiser");

	// initialize GL
	if( !initGL()) {
			return;
	}
	initShader();

	// register callbacks
	glutDisplayFunc( display);
	glutKeyboardFunc( keyboard);
	glutMouseFunc( mouse);
	glutMotionFunc( motion);
    
	// create VBO's
	createVBO( &sphereVerts, SPHERE_SLICES* (SPHERE_STACKS+1) * sizeof(glm::vec3));
	createVBO( &sphereNormals, SPHERE_SLICES* (SPHERE_STACKS+1) * sizeof (glm::vec3));
	setVertexBufferData();

	// create TBO
	createTBO( &LTin_ltin_random_movement_tbo, &LTin_ltin_random_movement_displacementTex, xmachine_memory_LTin_MAX * sizeof( glm::vec4));
	
	createTBO( &LTin_stable_contact_tbo, &LTin_stable_contact_displacementTex, xmachine_memory_LTin_MAX * sizeof( glm::vec4));
	
	createTBO( &LTin_localised_movement_tbo, &LTin_localised_movement_displacementTex, xmachine_memory_LTin_MAX * sizeof( glm::vec4));
	
	createTBO( &LTi_lti_random_movement_tbo, &LTi_lti_random_movement_displacementTex, xmachine_memory_LTi_MAX * sizeof( glm::vec4));
	
	createTBO( &LTi_responding_tbo, &LTi_responding_displacementTex, xmachine_memory_LTi_MAX * sizeof( glm::vec4));
	
	createTBO( &LTi_contact_tbo, &LTi_contact_displacementTex, xmachine_memory_LTi_MAX * sizeof( glm::vec4));
	
	createTBO( &LTi_adhesion_tbo, &LTi_adhesion_displacementTex, xmachine_memory_LTi_MAX * sizeof( glm::vec4));
	
	createTBO( &LTo_no_expression_tbo, &LTo_no_expression_displacementTex, xmachine_memory_LTo_MAX * sizeof( glm::vec4));
	
	createTBO( &LTo_expression_tbo, &LTo_expression_displacementTex, xmachine_memory_LTo_MAX * sizeof( glm::vec4));
	
	createTBO( &LTo_adhesion_upregulation_tbo, &LTo_adhesion_upregulation_displacementTex, xmachine_memory_LTo_MAX * sizeof( glm::vec4));
	
	createTBO( &LTo_chemokine_upregulation_tbo, &LTo_chemokine_upregulation_displacementTex, xmachine_memory_LTo_MAX * sizeof( glm::vec4));
	
	createTBO( &LTo_mature_tbo, &LTo_mature_displacementTex, xmachine_memory_LTo_MAX * sizeof( glm::vec4));
	
	createTBO( &LTo_downregulated_tbo, &LTo_downregulated_displacementTex, xmachine_memory_LTo_MAX * sizeof( glm::vec4));
	

	//set shader uniforms
	glUseProgram(shaderProgram);

	//create a events for timer
	hipEventCreate(&start);
	hipEventCreate(&stop);
}

void runVisualisation(){
	// start rendering mainloop
	glutMainLoop();
}

////////////////////////////////////////////////////////////////////////////////
//! Run the Cuda part of the computation
////////////////////////////////////////////////////////////////////////////////
void runCuda()
{
#ifdef SIMULATION_DELAY
	delay_count++;
	if (delay_count == SIMULATION_DELAY){
		delay_count = 0;
		singleIteration();
	}
#else
	singleIteration();
#endif

	//kernals sizes
	int threads_per_tile = 256;
	int tile_size;
	dim3 grid;
	dim3 threads;
	glm::vec3 centralise;

	//pointer
	glm::vec4 *dptr;

	
	if (get_agent_LTin_ltin_random_movement_count() > 0)
	{
		// map OpenGL buffer object for writing from CUDA
		gpuErrchk(cudaGLMapBufferObject( (void**)&dptr, LTin_ltin_random_movement_tbo));
		//cuda block size
		tile_size = (int) ceil((float)get_agent_LTin_ltin_random_movement_count()/threads_per_tile);
		grid = dim3(tile_size, 1, 1);
		threads = dim3(threads_per_tile, 1, 1);
        
        //continuous variables  
        centralise = getMaximumBounds() + getMinimumBounds();
        centralise /= 2;
        
		output_LTin_agent_to_VBO<<< grid, threads>>>(get_device_LTin_ltin_random_movement_agents(), dptr, centralise);
		gpuErrchkLaunch();
		// unmap buffer object
		gpuErrchk(cudaGLUnmapBufferObject(LTin_ltin_random_movement_tbo));
	}
	
	if (get_agent_LTin_stable_contact_count() > 0)
	{
		// map OpenGL buffer object for writing from CUDA
		gpuErrchk(cudaGLMapBufferObject( (void**)&dptr, LTin_stable_contact_tbo));
		//cuda block size
		tile_size = (int) ceil((float)get_agent_LTin_stable_contact_count()/threads_per_tile);
		grid = dim3(tile_size, 1, 1);
		threads = dim3(threads_per_tile, 1, 1);
        
        //continuous variables  
        centralise = getMaximumBounds() + getMinimumBounds();
        centralise /= 2;
        
		output_LTin_agent_to_VBO<<< grid, threads>>>(get_device_LTin_stable_contact_agents(), dptr, centralise);
		gpuErrchkLaunch();
		// unmap buffer object
		gpuErrchk(cudaGLUnmapBufferObject(LTin_stable_contact_tbo));
	}
	
	if (get_agent_LTin_localised_movement_count() > 0)
	{
		// map OpenGL buffer object for writing from CUDA
		gpuErrchk(cudaGLMapBufferObject( (void**)&dptr, LTin_localised_movement_tbo));
		//cuda block size
		tile_size = (int) ceil((float)get_agent_LTin_localised_movement_count()/threads_per_tile);
		grid = dim3(tile_size, 1, 1);
		threads = dim3(threads_per_tile, 1, 1);
        
        //continuous variables  
        centralise = getMaximumBounds() + getMinimumBounds();
        centralise /= 2;
        
		output_LTin_agent_to_VBO<<< grid, threads>>>(get_device_LTin_localised_movement_agents(), dptr, centralise);
		gpuErrchkLaunch();
		// unmap buffer object
		gpuErrchk(cudaGLUnmapBufferObject(LTin_localised_movement_tbo));
	}
	
	if (get_agent_LTi_lti_random_movement_count() > 0)
	{
		// map OpenGL buffer object for writing from CUDA
		gpuErrchk(cudaGLMapBufferObject( (void**)&dptr, LTi_lti_random_movement_tbo));
		//cuda block size
		tile_size = (int) ceil((float)get_agent_LTi_lti_random_movement_count()/threads_per_tile);
		grid = dim3(tile_size, 1, 1);
		threads = dim3(threads_per_tile, 1, 1);
        
        //continuous variables  
        centralise = getMaximumBounds() + getMinimumBounds();
        centralise /= 2;
        
		output_LTi_agent_to_VBO<<< grid, threads>>>(get_device_LTi_lti_random_movement_agents(), dptr, centralise);
		gpuErrchkLaunch();
		// unmap buffer object
		gpuErrchk(cudaGLUnmapBufferObject(LTi_lti_random_movement_tbo));
	}
	
	if (get_agent_LTi_responding_count() > 0)
	{
		// map OpenGL buffer object for writing from CUDA
		gpuErrchk(cudaGLMapBufferObject( (void**)&dptr, LTi_responding_tbo));
		//cuda block size
		tile_size = (int) ceil((float)get_agent_LTi_responding_count()/threads_per_tile);
		grid = dim3(tile_size, 1, 1);
		threads = dim3(threads_per_tile, 1, 1);
        
        //continuous variables  
        centralise = getMaximumBounds() + getMinimumBounds();
        centralise /= 2;
        
		output_LTi_agent_to_VBO<<< grid, threads>>>(get_device_LTi_responding_agents(), dptr, centralise);
		gpuErrchkLaunch();
		// unmap buffer object
		gpuErrchk(cudaGLUnmapBufferObject(LTi_responding_tbo));
	}
	
	if (get_agent_LTi_contact_count() > 0)
	{
		// map OpenGL buffer object for writing from CUDA
		gpuErrchk(cudaGLMapBufferObject( (void**)&dptr, LTi_contact_tbo));
		//cuda block size
		tile_size = (int) ceil((float)get_agent_LTi_contact_count()/threads_per_tile);
		grid = dim3(tile_size, 1, 1);
		threads = dim3(threads_per_tile, 1, 1);
        
        //continuous variables  
        centralise = getMaximumBounds() + getMinimumBounds();
        centralise /= 2;
        
		output_LTi_agent_to_VBO<<< grid, threads>>>(get_device_LTi_contact_agents(), dptr, centralise);
		gpuErrchkLaunch();
		// unmap buffer object
		gpuErrchk(cudaGLUnmapBufferObject(LTi_contact_tbo));
	}
	
	if (get_agent_LTi_adhesion_count() > 0)
	{
		// map OpenGL buffer object for writing from CUDA
		gpuErrchk(cudaGLMapBufferObject( (void**)&dptr, LTi_adhesion_tbo));
		//cuda block size
		tile_size = (int) ceil((float)get_agent_LTi_adhesion_count()/threads_per_tile);
		grid = dim3(tile_size, 1, 1);
		threads = dim3(threads_per_tile, 1, 1);
        
        //continuous variables  
        centralise = getMaximumBounds() + getMinimumBounds();
        centralise /= 2;
        
		output_LTi_agent_to_VBO<<< grid, threads>>>(get_device_LTi_adhesion_agents(), dptr, centralise);
		gpuErrchkLaunch();
		// unmap buffer object
		gpuErrchk(cudaGLUnmapBufferObject(LTi_adhesion_tbo));
	}
	
	if (get_agent_LTo_no_expression_count() > 0)
	{
		// map OpenGL buffer object for writing from CUDA
		gpuErrchk(cudaGLMapBufferObject( (void**)&dptr, LTo_no_expression_tbo));
		//cuda block size
		tile_size = (int) ceil((float)get_agent_LTo_no_expression_count()/threads_per_tile);
		grid = dim3(tile_size, 1, 1);
		threads = dim3(threads_per_tile, 1, 1);
        
        //continuous variables  
        centralise = getMaximumBounds() + getMinimumBounds();
        centralise /= 2;
        
		output_LTo_agent_to_VBO<<< grid, threads>>>(get_device_LTo_no_expression_agents(), dptr, centralise);
		gpuErrchkLaunch();
		// unmap buffer object
		gpuErrchk(cudaGLUnmapBufferObject(LTo_no_expression_tbo));
	}
	
	if (get_agent_LTo_expression_count() > 0)
	{
		// map OpenGL buffer object for writing from CUDA
		gpuErrchk(cudaGLMapBufferObject( (void**)&dptr, LTo_expression_tbo));
		//cuda block size
		tile_size = (int) ceil((float)get_agent_LTo_expression_count()/threads_per_tile);
		grid = dim3(tile_size, 1, 1);
		threads = dim3(threads_per_tile, 1, 1);
        
        //continuous variables  
        centralise = getMaximumBounds() + getMinimumBounds();
        centralise /= 2;
        
		output_LTo_agent_to_VBO<<< grid, threads>>>(get_device_LTo_expression_agents(), dptr, centralise);
		gpuErrchkLaunch();
		// unmap buffer object
		gpuErrchk(cudaGLUnmapBufferObject(LTo_expression_tbo));
	}
	
	if (get_agent_LTo_adhesion_upregulation_count() > 0)
	{
		// map OpenGL buffer object for writing from CUDA
		gpuErrchk(cudaGLMapBufferObject( (void**)&dptr, LTo_adhesion_upregulation_tbo));
		//cuda block size
		tile_size = (int) ceil((float)get_agent_LTo_adhesion_upregulation_count()/threads_per_tile);
		grid = dim3(tile_size, 1, 1);
		threads = dim3(threads_per_tile, 1, 1);
        
        //continuous variables  
        centralise = getMaximumBounds() + getMinimumBounds();
        centralise /= 2;
        
		output_LTo_agent_to_VBO<<< grid, threads>>>(get_device_LTo_adhesion_upregulation_agents(), dptr, centralise);
		gpuErrchkLaunch();
		// unmap buffer object
		gpuErrchk(cudaGLUnmapBufferObject(LTo_adhesion_upregulation_tbo));
	}
	
	if (get_agent_LTo_chemokine_upregulation_count() > 0)
	{
		// map OpenGL buffer object for writing from CUDA
		gpuErrchk(cudaGLMapBufferObject( (void**)&dptr, LTo_chemokine_upregulation_tbo));
		//cuda block size
		tile_size = (int) ceil((float)get_agent_LTo_chemokine_upregulation_count()/threads_per_tile);
		grid = dim3(tile_size, 1, 1);
		threads = dim3(threads_per_tile, 1, 1);
        
        //continuous variables  
        centralise = getMaximumBounds() + getMinimumBounds();
        centralise /= 2;
        
		output_LTo_agent_to_VBO<<< grid, threads>>>(get_device_LTo_chemokine_upregulation_agents(), dptr, centralise);
		gpuErrchkLaunch();
		// unmap buffer object
		gpuErrchk(cudaGLUnmapBufferObject(LTo_chemokine_upregulation_tbo));
	}
	
	if (get_agent_LTo_mature_count() > 0)
	{
		// map OpenGL buffer object for writing from CUDA
		gpuErrchk(cudaGLMapBufferObject( (void**)&dptr, LTo_mature_tbo));
		//cuda block size
		tile_size = (int) ceil((float)get_agent_LTo_mature_count()/threads_per_tile);
		grid = dim3(tile_size, 1, 1);
		threads = dim3(threads_per_tile, 1, 1);
        
        //continuous variables  
        centralise = getMaximumBounds() + getMinimumBounds();
        centralise /= 2;
        
		output_LTo_agent_to_VBO<<< grid, threads>>>(get_device_LTo_mature_agents(), dptr, centralise);
		gpuErrchkLaunch();
		// unmap buffer object
		gpuErrchk(cudaGLUnmapBufferObject(LTo_mature_tbo));
	}
	
	if (get_agent_LTo_downregulated_count() > 0)
	{
		// map OpenGL buffer object for writing from CUDA
		gpuErrchk(cudaGLMapBufferObject( (void**)&dptr, LTo_downregulated_tbo));
		//cuda block size
		tile_size = (int) ceil((float)get_agent_LTo_downregulated_count()/threads_per_tile);
		grid = dim3(tile_size, 1, 1);
		threads = dim3(threads_per_tile, 1, 1);
        
        //continuous variables  
        centralise = getMaximumBounds() + getMinimumBounds();
        centralise /= 2;
        
		output_LTo_agent_to_VBO<<< grid, threads>>>(get_device_LTo_downregulated_agents(), dptr, centralise);
		gpuErrchkLaunch();
		// unmap buffer object
		gpuErrchk(cudaGLUnmapBufferObject(LTo_downregulated_tbo));
	}
	
}

////////////////////////////////////////////////////////////////////////////////
//! Initialize GL
////////////////////////////////////////////////////////////////////////////////
int initGL()
{
	// initialize necessary OpenGL extensions
	glewInit();
	if (! glewIsSupported( "GL_VERSION_2_0 " 
		"GL_ARB_pixel_buffer_object")) {
		fprintf( stderr, "ERROR: Support for necessary OpenGL extensions missing.");
		fflush( stderr);
		return 1;
	}

	// default initialization
	glClearColor( 1.0, 1.0, 1.0, 1.0);
	glEnable( GL_DEPTH_TEST);

	// viewport
	glViewport( 0, 0, WINDOW_WIDTH, WINDOW_HEIGHT);

	// projection
	glMatrixMode( GL_PROJECTION);
	glLoadIdentity();
	gluPerspective(45.0, (GLfloat)WINDOW_WIDTH / (GLfloat) WINDOW_HEIGHT, NEAR_CLIP, FAR_CLIP);

	checkGLError();

	//lighting
	glEnable(GL_LIGHTING);
	glEnable(GL_LIGHT0);

	return 1;
}

////////////////////////////////////////////////////////////////////////////////
//! Initialize GLSL Vertex Shader
////////////////////////////////////////////////////////////////////////////////
void initShader()
{
	const char* v = vertexShaderSource;
	const char* f = fragmentShaderSource;

	//vertex shader
	vertexShader = glCreateShader(GL_VERTEX_SHADER);
	glShaderSource(vertexShader, 1, &v, 0);
	glCompileShader(vertexShader);

	//fragment shader
	fragmentShader = glCreateShader(GL_FRAGMENT_SHADER);
	glShaderSource(fragmentShader, 1, &f, 0);
	glCompileShader(fragmentShader);

	//program
	shaderProgram = glCreateProgram();
	glAttachShader(shaderProgram, vertexShader);
	glAttachShader(shaderProgram, fragmentShader);
	glLinkProgram(shaderProgram);

	// check for errors
	GLint status;
	glGetShaderiv(vertexShader, GL_COMPILE_STATUS, &status);
	if (status == GL_FALSE){
		printf("ERROR: Shader Compilation Error\n");
		char data[262144];
		int len;
		glGetShaderInfoLog(vertexShader, 262144, &len, data); 
		printf("%s", data);
	}
	glGetShaderiv(fragmentShader, GL_COMPILE_STATUS, &status);
	if (status == GL_FALSE){
		printf("ERROR: Shader Compilation Error\n");
		char data[262144];
		int len;
		glGetShaderInfoLog(fragmentShader, 262144, &len, data); 
		printf("%s", data);
	}
	glGetProgramiv(shaderProgram, GL_LINK_STATUS, &status);
	if (status == GL_FALSE){
		printf("ERROR: Shader Program Link Error\n");
	}

	// get shader variables
	vs_displacementMap = glGetUniformLocation(shaderProgram, "displacementMap");
	vs_mapIndex = glGetAttribLocation(shaderProgram, "mapIndex"); 
}

////////////////////////////////////////////////////////////////////////////////
//! Create VBO
////////////////////////////////////////////////////////////////////////////////
void createVBO(GLuint* vbo, GLuint size)
{
	// create buffer object
	glGenBuffers( 1, vbo);
	glBindBuffer( GL_ARRAY_BUFFER, *vbo);

	// initialize buffer object
	glBufferData( GL_ARRAY_BUFFER, size, 0, GL_STATIC_DRAW);

	glBindBuffer( GL_ARRAY_BUFFER, 0);

	checkGLError();
}

////////////////////////////////////////////////////////////////////////////////
//! Delete VBO
////////////////////////////////////////////////////////////////////////////////
void deleteVBO( GLuint* vbo)
{
	glBindBuffer( 1, *vbo);
	glDeleteBuffers( 1, vbo);

	*vbo = 0;
}

////////////////////////////////////////////////////////////////////////////////
//! Create TBO
////////////////////////////////////////////////////////////////////////////////
void createTBO(GLuint* tbo, GLuint* tex, GLuint size)
{
	// create buffer object
	glGenBuffers( 1, tbo);
	glBindBuffer( GL_TEXTURE_BUFFER_EXT, *tbo);

	// initialize buffer object
	glBufferData( GL_TEXTURE_BUFFER_EXT, size, 0, GL_DYNAMIC_DRAW);

	//tex
	glGenTextures(1, tex);
	glBindTexture(GL_TEXTURE_BUFFER_EXT, *tex);
	glTexBufferEXT(GL_TEXTURE_BUFFER_EXT, GL_RGBA32F_ARB, *tbo); 
	glBindBuffer(GL_TEXTURE_BUFFER_EXT, 0);

    // register buffer object with CUDA
    gpuErrchk(cudaGLRegisterBufferObject(*tbo));

    checkGLError();
}

////////////////////////////////////////////////////////////////////////////////
//! Delete TBO
////////////////////////////////////////////////////////////////////////////////
void deleteTBO( GLuint* tbo)
{
	glBindBuffer( 1, *tbo);
	glDeleteBuffers( 1, tbo);

	gpuErrchk(cudaGLUnregisterBufferObject(*tbo));

	*tbo = 0;
}

////////////////////////////////////////////////////////////////////////////////
//! Set Sphere Vertex Data
////////////////////////////////////////////////////////////////////////////////

static void setSphereVertex(glm::vec3* data, int slice, int stack) {
	float PI = 3.14159265358;
    
	double sl = 2*PI*slice/SPHERE_SLICES;
	double st = 2*PI*stack/SPHERE_STACKS;
 
	data->x = cos(st)*sin(sl) * SPHERE_RADIUS;
	data->y = sin(st)*sin(sl) * SPHERE_RADIUS;
	data->z = cos(sl) * SPHERE_RADIUS;
}


////////////////////////////////////////////////////////////////////////////////
//! Set Sphere Normal Data
////////////////////////////////////////////////////////////////////////////////

static void setSphereNormal(glm::vec3* data, int slice, int stack) {
	float PI = 3.14159265358;
    
	double sl = 2*PI*slice/SPHERE_SLICES;
	double st = 2*PI*stack/SPHERE_STACKS;
 
	data->x = cos(st)*sin(sl);
	data->y = sin(st)*sin(sl);
	data->z = cos(sl);
}


////////////////////////////////////////////////////////////////////////////////
//! Set Vertex Buffer Data
////////////////////////////////////////////////////////////////////////////////
void setVertexBufferData()
{
	int slice, stack;
	int i;

	// upload vertex points data
	glBindBuffer(GL_ARRAY_BUFFER, sphereVerts);
	glm::vec3* verts =( glm::vec3*)glMapBuffer(GL_ARRAY_BUFFER, GL_WRITE_ONLY);
	i = 0;
	for (slice=0; slice<SPHERE_SLICES/2; slice++) {
		for (stack=0; stack<=SPHERE_STACKS; stack++) {
			setSphereVertex(&verts[i++], slice, stack);
			setSphereVertex(&verts[i++], slice+1, stack);
		}
    }
	glUnmapBuffer(GL_ARRAY_BUFFER);

	// upload vertex normal data
	glBindBuffer(GL_ARRAY_BUFFER, sphereNormals);
	glm::vec3* normals =( glm::vec3*)glMapBuffer(GL_ARRAY_BUFFER, GL_WRITE_ONLY);
	i = 0;
	for (slice=0; slice<SPHERE_SLICES/2; slice++) {
		for (stack=0; stack<=SPHERE_STACKS; stack++) {
			setSphereNormal(&normals[i++], slice, stack);
			setSphereNormal(&normals[i++], slice+1, stack);
		}
    }
	glUnmapBuffer(GL_ARRAY_BUFFER);
}


////////////////////////////////////////////////////////////////////////////////
//! Display callback
////////////////////////////////////////////////////////////////////////////////
void display()
{
	float millis;
	
	//CUDA start Timing
	hipEventRecord(start);

	// run CUDA kernel to generate vertex positions
	runCuda();

	glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);

	// set view matrix
	glMatrixMode(GL_MODELVIEW);
	glLoadIdentity();


	//zoom
	glTranslatef(0.0, 0.0, translate_z); 
	//move
	glRotatef(rotate_x, 1.0, 0.0, 0.0);
	glRotatef(rotate_y, 0.0, 0.0, 1.0);


	//Set light position
	glLightfv(GL_LIGHT0, GL_POSITION, LIGHT_POSITION);

	
	//Draw LTin Agents in ltin_random_movement state
	glActiveTexture(GL_TEXTURE0);
	glBindTexture(GL_TEXTURE_BUFFER_EXT, LTin_ltin_random_movement_displacementTex);
	//loop
	for (int i=0; i< get_agent_LTin_ltin_random_movement_count(); i++){
		glVertexAttrib1f(vs_mapIndex, (float)i);
		
		//draw using vertex and attribute data on the gpu (fast)
		glEnableClientState(GL_VERTEX_ARRAY);
		glEnableClientState(GL_NORMAL_ARRAY);

		glBindBuffer(GL_ARRAY_BUFFER, sphereVerts);
		glVertexPointer(3, GL_FLOAT, 0, 0);

		glBindBuffer(GL_ARRAY_BUFFER, sphereNormals);
		glNormalPointer(GL_FLOAT, 0, 0);

		glDrawArrays(GL_TRIANGLE_STRIP, 0, SPHERE_SLICES * (SPHERE_STACKS+1));

		glDisableClientState(GL_NORMAL_ARRAY);
		glDisableClientState(GL_VERTEX_ARRAY);
	}
	
	//Draw LTin Agents in stable_contact state
	glActiveTexture(GL_TEXTURE0);
	glBindTexture(GL_TEXTURE_BUFFER_EXT, LTin_stable_contact_displacementTex);
	//loop
	for (int i=0; i< get_agent_LTin_stable_contact_count(); i++){
		glVertexAttrib1f(vs_mapIndex, (float)i);
		
		//draw using vertex and attribute data on the gpu (fast)
		glEnableClientState(GL_VERTEX_ARRAY);
		glEnableClientState(GL_NORMAL_ARRAY);

		glBindBuffer(GL_ARRAY_BUFFER, sphereVerts);
		glVertexPointer(3, GL_FLOAT, 0, 0);

		glBindBuffer(GL_ARRAY_BUFFER, sphereNormals);
		glNormalPointer(GL_FLOAT, 0, 0);

		glDrawArrays(GL_TRIANGLE_STRIP, 0, SPHERE_SLICES * (SPHERE_STACKS+1));

		glDisableClientState(GL_NORMAL_ARRAY);
		glDisableClientState(GL_VERTEX_ARRAY);
	}
	
	//Draw LTin Agents in localised_movement state
	glActiveTexture(GL_TEXTURE0);
	glBindTexture(GL_TEXTURE_BUFFER_EXT, LTin_localised_movement_displacementTex);
	//loop
	for (int i=0; i< get_agent_LTin_localised_movement_count(); i++){
		glVertexAttrib1f(vs_mapIndex, (float)i);
		
		//draw using vertex and attribute data on the gpu (fast)
		glEnableClientState(GL_VERTEX_ARRAY);
		glEnableClientState(GL_NORMAL_ARRAY);

		glBindBuffer(GL_ARRAY_BUFFER, sphereVerts);
		glVertexPointer(3, GL_FLOAT, 0, 0);

		glBindBuffer(GL_ARRAY_BUFFER, sphereNormals);
		glNormalPointer(GL_FLOAT, 0, 0);

		glDrawArrays(GL_TRIANGLE_STRIP, 0, SPHERE_SLICES * (SPHERE_STACKS+1));

		glDisableClientState(GL_NORMAL_ARRAY);
		glDisableClientState(GL_VERTEX_ARRAY);
	}
	
	//Draw LTi Agents in lti_random_movement state
	glActiveTexture(GL_TEXTURE0);
	glBindTexture(GL_TEXTURE_BUFFER_EXT, LTi_lti_random_movement_displacementTex);
	//loop
	for (int i=0; i< get_agent_LTi_lti_random_movement_count(); i++){
		glVertexAttrib1f(vs_mapIndex, (float)i);
		
		//draw using vertex and attribute data on the gpu (fast)
		glEnableClientState(GL_VERTEX_ARRAY);
		glEnableClientState(GL_NORMAL_ARRAY);

		glBindBuffer(GL_ARRAY_BUFFER, sphereVerts);
		glVertexPointer(3, GL_FLOAT, 0, 0);

		glBindBuffer(GL_ARRAY_BUFFER, sphereNormals);
		glNormalPointer(GL_FLOAT, 0, 0);

		glDrawArrays(GL_TRIANGLE_STRIP, 0, SPHERE_SLICES * (SPHERE_STACKS+1));

		glDisableClientState(GL_NORMAL_ARRAY);
		glDisableClientState(GL_VERTEX_ARRAY);
	}
	
	//Draw LTi Agents in responding state
	glActiveTexture(GL_TEXTURE0);
	glBindTexture(GL_TEXTURE_BUFFER_EXT, LTi_responding_displacementTex);
	//loop
	for (int i=0; i< get_agent_LTi_responding_count(); i++){
		glVertexAttrib1f(vs_mapIndex, (float)i);
		
		//draw using vertex and attribute data on the gpu (fast)
		glEnableClientState(GL_VERTEX_ARRAY);
		glEnableClientState(GL_NORMAL_ARRAY);

		glBindBuffer(GL_ARRAY_BUFFER, sphereVerts);
		glVertexPointer(3, GL_FLOAT, 0, 0);

		glBindBuffer(GL_ARRAY_BUFFER, sphereNormals);
		glNormalPointer(GL_FLOAT, 0, 0);

		glDrawArrays(GL_TRIANGLE_STRIP, 0, SPHERE_SLICES * (SPHERE_STACKS+1));

		glDisableClientState(GL_NORMAL_ARRAY);
		glDisableClientState(GL_VERTEX_ARRAY);
	}
	
	//Draw LTi Agents in contact state
	glActiveTexture(GL_TEXTURE0);
	glBindTexture(GL_TEXTURE_BUFFER_EXT, LTi_contact_displacementTex);
	//loop
	for (int i=0; i< get_agent_LTi_contact_count(); i++){
		glVertexAttrib1f(vs_mapIndex, (float)i);
		
		//draw using vertex and attribute data on the gpu (fast)
		glEnableClientState(GL_VERTEX_ARRAY);
		glEnableClientState(GL_NORMAL_ARRAY);

		glBindBuffer(GL_ARRAY_BUFFER, sphereVerts);
		glVertexPointer(3, GL_FLOAT, 0, 0);

		glBindBuffer(GL_ARRAY_BUFFER, sphereNormals);
		glNormalPointer(GL_FLOAT, 0, 0);

		glDrawArrays(GL_TRIANGLE_STRIP, 0, SPHERE_SLICES * (SPHERE_STACKS+1));

		glDisableClientState(GL_NORMAL_ARRAY);
		glDisableClientState(GL_VERTEX_ARRAY);
	}
	
	//Draw LTi Agents in adhesion state
	glActiveTexture(GL_TEXTURE0);
	glBindTexture(GL_TEXTURE_BUFFER_EXT, LTi_adhesion_displacementTex);
	//loop
	for (int i=0; i< get_agent_LTi_adhesion_count(); i++){
		glVertexAttrib1f(vs_mapIndex, (float)i);
		
		//draw using vertex and attribute data on the gpu (fast)
		glEnableClientState(GL_VERTEX_ARRAY);
		glEnableClientState(GL_NORMAL_ARRAY);

		glBindBuffer(GL_ARRAY_BUFFER, sphereVerts);
		glVertexPointer(3, GL_FLOAT, 0, 0);

		glBindBuffer(GL_ARRAY_BUFFER, sphereNormals);
		glNormalPointer(GL_FLOAT, 0, 0);

		glDrawArrays(GL_TRIANGLE_STRIP, 0, SPHERE_SLICES * (SPHERE_STACKS+1));

		glDisableClientState(GL_NORMAL_ARRAY);
		glDisableClientState(GL_VERTEX_ARRAY);
	}
	
	//Draw LTo Agents in no_expression state
	glActiveTexture(GL_TEXTURE0);
	glBindTexture(GL_TEXTURE_BUFFER_EXT, LTo_no_expression_displacementTex);
	//loop
	for (int i=0; i< get_agent_LTo_no_expression_count(); i++){
		glVertexAttrib1f(vs_mapIndex, (float)i);
		
		//draw using vertex and attribute data on the gpu (fast)
		glEnableClientState(GL_VERTEX_ARRAY);
		glEnableClientState(GL_NORMAL_ARRAY);

		glBindBuffer(GL_ARRAY_BUFFER, sphereVerts);
		glVertexPointer(3, GL_FLOAT, 0, 0);

		glBindBuffer(GL_ARRAY_BUFFER, sphereNormals);
		glNormalPointer(GL_FLOAT, 0, 0);

		glDrawArrays(GL_TRIANGLE_STRIP, 0, SPHERE_SLICES * (SPHERE_STACKS+1));

		glDisableClientState(GL_NORMAL_ARRAY);
		glDisableClientState(GL_VERTEX_ARRAY);
	}
	
	//Draw LTo Agents in expression state
	glActiveTexture(GL_TEXTURE0);
	glBindTexture(GL_TEXTURE_BUFFER_EXT, LTo_expression_displacementTex);
	//loop
	for (int i=0; i< get_agent_LTo_expression_count(); i++){
		glVertexAttrib1f(vs_mapIndex, (float)i);
		
		//draw using vertex and attribute data on the gpu (fast)
		glEnableClientState(GL_VERTEX_ARRAY);
		glEnableClientState(GL_NORMAL_ARRAY);

		glBindBuffer(GL_ARRAY_BUFFER, sphereVerts);
		glVertexPointer(3, GL_FLOAT, 0, 0);

		glBindBuffer(GL_ARRAY_BUFFER, sphereNormals);
		glNormalPointer(GL_FLOAT, 0, 0);

		glDrawArrays(GL_TRIANGLE_STRIP, 0, SPHERE_SLICES * (SPHERE_STACKS+1));

		glDisableClientState(GL_NORMAL_ARRAY);
		glDisableClientState(GL_VERTEX_ARRAY);
	}
	
	//Draw LTo Agents in adhesion_upregulation state
	glActiveTexture(GL_TEXTURE0);
	glBindTexture(GL_TEXTURE_BUFFER_EXT, LTo_adhesion_upregulation_displacementTex);
	//loop
	for (int i=0; i< get_agent_LTo_adhesion_upregulation_count(); i++){
		glVertexAttrib1f(vs_mapIndex, (float)i);
		
		//draw using vertex and attribute data on the gpu (fast)
		glEnableClientState(GL_VERTEX_ARRAY);
		glEnableClientState(GL_NORMAL_ARRAY);

		glBindBuffer(GL_ARRAY_BUFFER, sphereVerts);
		glVertexPointer(3, GL_FLOAT, 0, 0);

		glBindBuffer(GL_ARRAY_BUFFER, sphereNormals);
		glNormalPointer(GL_FLOAT, 0, 0);

		glDrawArrays(GL_TRIANGLE_STRIP, 0, SPHERE_SLICES * (SPHERE_STACKS+1));

		glDisableClientState(GL_NORMAL_ARRAY);
		glDisableClientState(GL_VERTEX_ARRAY);
	}
	
	//Draw LTo Agents in chemokine_upregulation state
	glActiveTexture(GL_TEXTURE0);
	glBindTexture(GL_TEXTURE_BUFFER_EXT, LTo_chemokine_upregulation_displacementTex);
	//loop
	for (int i=0; i< get_agent_LTo_chemokine_upregulation_count(); i++){
		glVertexAttrib1f(vs_mapIndex, (float)i);
		
		//draw using vertex and attribute data on the gpu (fast)
		glEnableClientState(GL_VERTEX_ARRAY);
		glEnableClientState(GL_NORMAL_ARRAY);

		glBindBuffer(GL_ARRAY_BUFFER, sphereVerts);
		glVertexPointer(3, GL_FLOAT, 0, 0);

		glBindBuffer(GL_ARRAY_BUFFER, sphereNormals);
		glNormalPointer(GL_FLOAT, 0, 0);

		glDrawArrays(GL_TRIANGLE_STRIP, 0, SPHERE_SLICES * (SPHERE_STACKS+1));

		glDisableClientState(GL_NORMAL_ARRAY);
		glDisableClientState(GL_VERTEX_ARRAY);
	}
	
	//Draw LTo Agents in mature state
	glActiveTexture(GL_TEXTURE0);
	glBindTexture(GL_TEXTURE_BUFFER_EXT, LTo_mature_displacementTex);
	//loop
	for (int i=0; i< get_agent_LTo_mature_count(); i++){
		glVertexAttrib1f(vs_mapIndex, (float)i);
		
		//draw using vertex and attribute data on the gpu (fast)
		glEnableClientState(GL_VERTEX_ARRAY);
		glEnableClientState(GL_NORMAL_ARRAY);

		glBindBuffer(GL_ARRAY_BUFFER, sphereVerts);
		glVertexPointer(3, GL_FLOAT, 0, 0);

		glBindBuffer(GL_ARRAY_BUFFER, sphereNormals);
		glNormalPointer(GL_FLOAT, 0, 0);

		glDrawArrays(GL_TRIANGLE_STRIP, 0, SPHERE_SLICES * (SPHERE_STACKS+1));

		glDisableClientState(GL_NORMAL_ARRAY);
		glDisableClientState(GL_VERTEX_ARRAY);
	}
	
	//Draw LTo Agents in downregulated state
	glActiveTexture(GL_TEXTURE0);
	glBindTexture(GL_TEXTURE_BUFFER_EXT, LTo_downregulated_displacementTex);
	//loop
	for (int i=0; i< get_agent_LTo_downregulated_count(); i++){
		glVertexAttrib1f(vs_mapIndex, (float)i);
		
		//draw using vertex and attribute data on the gpu (fast)
		glEnableClientState(GL_VERTEX_ARRAY);
		glEnableClientState(GL_NORMAL_ARRAY);

		glBindBuffer(GL_ARRAY_BUFFER, sphereVerts);
		glVertexPointer(3, GL_FLOAT, 0, 0);

		glBindBuffer(GL_ARRAY_BUFFER, sphereNormals);
		glNormalPointer(GL_FLOAT, 0, 0);

		glDrawArrays(GL_TRIANGLE_STRIP, 0, SPHERE_SLICES * (SPHERE_STACKS+1));

		glDisableClientState(GL_NORMAL_ARRAY);
		glDisableClientState(GL_VERTEX_ARRAY);
	}
	

	//CUDA stop timing
	hipEventRecord(stop);
	glFlush();
	hipEventSynchronize(stop);
	hipEventElapsedTime(&millis, start, stop);
  frame_time += millis;

	if(frame_count == display_rate){
		char title [100];
		sprintf(title, "Execution & Rendering Total: %f (FPS), %f milliseconds per frame", display_rate/(frame_time/1000.0f), frame_time/display_rate);
		glutSetWindowTitle(title);

		//reset
		frame_count = 0;
    frame_time = 0.0;
	}else{
		frame_count++;
	}


	glutSwapBuffers();
	glutPostRedisplay();

}

////////////////////////////////////////////////////////////////////////////////
//! Keyboard events handler
////////////////////////////////////////////////////////////////////////////////
void keyboard( unsigned char key, int /*x*/, int /*y*/)
{
	switch( key) {
	case( 27) :
		deleteVBO( &sphereVerts);
		deleteVBO( &sphereNormals);
		
		deleteTBO( &LTin_ltin_random_movement_tbo);
		
		deleteTBO( &LTin_stable_contact_tbo);
		
		deleteTBO( &LTin_localised_movement_tbo);
		
		deleteTBO( &LTi_lti_random_movement_tbo);
		
		deleteTBO( &LTi_responding_tbo);
		
		deleteTBO( &LTi_contact_tbo);
		
		deleteTBO( &LTi_adhesion_tbo);
		
		deleteTBO( &LTo_no_expression_tbo);
		
		deleteTBO( &LTo_expression_tbo);
		
		deleteTBO( &LTo_adhesion_upregulation_tbo);
		
		deleteTBO( &LTo_chemokine_upregulation_tbo);
		
		deleteTBO( &LTo_mature_tbo);
		
		deleteTBO( &LTo_downregulated_tbo);
		
		hipEventDestroy(start);
    hipEventDestroy(stop);
		exit( 0);
	}
}

////////////////////////////////////////////////////////////////////////////////
//! Mouse event handlers
////////////////////////////////////////////////////////////////////////////////
void mouse(int button, int state, int x, int y)
{
	if (state == GLUT_DOWN) {
		mouse_buttons |= 1<<button;
	} else if (state == GLUT_UP) {
		mouse_buttons = 0;
	}

	mouse_old_x = x;
	mouse_old_y = y;
	glutPostRedisplay();
}

void motion(int x, int y)
{
	float dx, dy;
	dx = x - mouse_old_x;
	dy = y - mouse_old_y;

	if (mouse_buttons & 1) {
		rotate_x += dy * 0.2;
		rotate_y += dx * 0.2;
	} else if (mouse_buttons & 4) {
		translate_z += dy * VIEW_DISTANCE * 0.001;
	}

  mouse_old_x = x;
  mouse_old_y = y;
}

void checkGLError(){
  int Error;
  if((Error = glGetError()) != GL_NO_ERROR)
  {
    const char* Message = (const char*)gluErrorString(Error);
    fprintf(stderr, "OpenGL Error : %s\n", Message);
  }
}
